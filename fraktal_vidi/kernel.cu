
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <windows.h>
#include "Fractal.h"
#include "CsikiFractal.h"

int main()
{
    drawFractal<CsikiFractal> (24, 20, 400, 500);
    return 0;
}
